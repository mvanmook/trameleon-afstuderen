#include "hip/hip_runtime.h"
/* ------------------------------------------------------- */

#include <tra/modules/nvidia/nvidia-converter-kernels.h>
#include <tra/log.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

/* ------------------------------------------------------- */

/*

  GENERAL INFO:

    The kernel that will resize a NV12 buffer. This is based on
    [this][0] and [this][1] example. This code is similar to most
    scalers I've found online. It uses texture memory and the
    `tex2D()` function to perform the scaling; I haven't found
    any clear documentation about why `tex2D()` is used, but it's
    most likely used take advantage of hardware interpolation.

  TODO:

    Although the performance is already amazing (2.8us on
    2080RTX, 4.5us on 1650GTX to resize 1920x1080 to 960x540), we
    might be able to optimize this a bit further by making the
    memory writes 32bit wide instead of 16bit. E.g. we are
    writing `uchar2` and we could potentially write
    `uchar4`. Though if we would like to use 32 bit wide writes,
    we should create two kernels; one for the luma and one for
    the chroma.

    Also, currently we don't use all arguments; currently there
    is not really a reason to keep them, though I might want to
    improve the kernel at some point which is why I keep them.
    
  REFERENCES:

    [0]: https://github.com/NVIDIA/cuda-samples/blob/master/Samples/5_Domain_Specific/NV12toBGRandResize/nv12_resize.cu
    [1]: https://github.com/NVIDIA/video-sdk-samples/blob/master/Samples/Utils/Resize.cu "NVIDIA Video SDK samples."

 */
__global__ static void cuda_resize_nv12_kernel(
  hipTextureObject_t inputTexY,
  hipTextureObject_t inputTexUv,
  uint32_t inputWidth,
  uint32_t inputHeight,
  uint32_t inputPitch,
  uint8_t* outputY,
  uint8_t* outputUv,
  uint32_t outputWidth,
  uint32_t outputHeight,
  uint32_t outputPitch,
  float scaleX,
  float scaleY
)
{

  /*
    Calculate the (i,j) coordinate where i and j will/should
    never exceed the `(outputWidth / 2)` and `(outputHeight /2)`
    as this kernel is supposed to run over the chroma plane: note
    that we use i,j directly to fetch the chroma samples below with
    the `tex2D()` call. 
  */
  uint32_t i = (blockIdx.x * blockDim.x) + threadIdx.x;
  uint32_t j = (blockIdx.y * blockDim.y) + threadIdx.y;

  /*
    @todo Are we only executing this kernel over all chroma
    pixels and are we therefore fetching and storing twice from
    the luma plane? 
  */
  uint32_t px = i * 2;
  uint32_t py = j * 2;

  /* Out of bounds? */
  if ((px + 1) >= outputWidth
      || (py + 1) >= outputHeight)
    {
       return;
    }

  uint8_t* p = outputY + (py * outputPitch) + px;

  /* Fetch 2 lines from luma plane and store in the resized buffer. */
  *(uchar2*) p = make_uchar2(
    tex2D<uint8_t>(inputTexY, scaleX * (px + 0), scaleY * (py + 0)),
    tex2D<uint8_t>(inputTexY, scaleX * (px + 1), scaleY * (py + 0))
  );
  
  *(uchar2*)(p + outputPitch) = make_uchar2(
    tex2D<uint8_t>(inputTexY, scaleX * (px + 0), scaleY * (py + 0)),
    tex2D<uint8_t>(inputTexY, scaleX * (px + 1), scaleY * (py + 1))
  );

  /* Fetch from chrome plane and store in the resized buffer. */
  p = outputUv + (j * outputPitch) + px;
  
  *(uchar2*)p = tex2D<uchar2>(
    inputTexUv,
    scaleX * i,
    scaleY * j 
  );
}

/* ------------------------------------------------------- */

/*

  GENERAL INFO:
  
    This is an experimental scaling function that resizes NV12
    input to the given output resolution. We assume that you have
    allocated the output buffer as a contiguous block of memory
    that can hold the NV12 buffer.

    This code is based on [nv12_resize.cu][0] and the [Resize.cu][1]
    samples which are pretty similar, though the version from the
    [cuda samples][0] is a bit more readable.

    We first create a `hipResourceDesc` that describes the input
    data, that we then use to create a `hipTextureObject_t`
    object for the Y-plane and another one for the UV-plane. The
    Y-plane texture read mode uses `hipReadModeElementType`
    which reads the element as its "original" value. The UV-plane
    texture uses `uchar2`, e.g. 2 channel data.

  GRID AND BLOCK SIZES

    The grid and block sizes determine how the kernel will
    operate on the data; It's important to use the right
    values. As this is the first time touching CUDA I'm
    using the values from the NVIDIA Video SDK [Samples][3]

  REFERENCES:

    [0]: https://github.com/NVIDIA/cuda-samples/blob/master/Samples/5_Domain_Specific/NV12toBGRandResize/nv12_resize.cu
    [1]: https://github.com/NVIDIA/video-sdk-samples/blob/master/Samples/Utils/Resize.cu "NVIDIA Video SDK samples."
    [2]: https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#texture-object-api "Texture Object API"
    [3]: https://github.com/NVIDIA/video-sdk-samples "NVIDIA Video SDK Samples"
    [4]: https://www.nvidia.com/content/GTC-2010/pdfs/2238_GTC2010.pdf "Better performance at lower occupancy"
    
 */
int tra_cuda_resize_nv12(
  void* inputDevicePtr,
  uint32_t inputWidth,
  uint32_t inputHeight,
  uint32_t inputPitch,
  void* outputDevicePtr,
  uint32_t outputWidth,
  uint32_t outputHeight,
  uint32_t outputPitch
)
{
  hipResourceDesc res_desc = { };
  hipTextureDesc tex_desc = { };
  hipTextureObject_t tex_y = 0;
  hipTextureObject_t tex_uv = 0;
  hipError_t status = hipSuccess;
  uint8_t* out_ptr_uv = NULL; /* Pointer to the UV-plane of the device buffer. */
  uint8_t* out_ptr_y = NULL;  /* (Casted) pointer to the Y-plane of the device buffer. */
  uint8_t* in_ptr_uv = NULL; /* Pointer to the UV-plane of the input data that we will resize. */
  uint8_t* in_ptr_y = NULL; /* Pointer to the Y-plane of the input data that we will resize. */
  float scale_x = 0.0f;
  float scale_y = 0.0f;
  dim3 block_size;
  dim3 grid_size;
  int r = 0;

  /* ------------------------------------------- */
  /* Validate input                              */
  /* ------------------------------------------- */
  
  if (NULL == inputDevicePtr) {
    TRAE("Cannot scale the NV12 buffer as the given input device pointer is NULL.");
    r = -10;
    goto error;
  }

  if (0 == inputWidth) {
    TRAE("Cannot scale the NV12 buffer as the given `inputWidth` is 0.");
    r = -20;
    goto error;
  }

  if (0 == inputHeight) {
    TRAE("Cannot scale the NV12 buffer as the given `inputHeight` is 0.");
    r = -30;
    goto error;
  }

  if (0 == inputPitch) {
    TRAE("Cannot scale the NV12 buffer as the given `inputPitch` is 0.");
    r = -40;
    goto error;
  }

  if (NULL == outputDevicePtr) {
    TRAE("Cannot scale the NV12 buffer as the given output device pointer is NULL.");
    r = -50;
    goto error;
  }

  if (0 == outputWidth) {
    TRAE("Cannot scale the NV12 buffer as the given `outputWidth` is 0.");
    r = -60;
    goto error;
  }

  if (0 == outputHeight) {
    TRAE("Cannot scale the NV12 buffer as the given `outputHeight` is 0.");
    r = -60;
    goto error;
  }

  if (0 == outputPitch) {
    TRAE("Cannot scale the NV12 buffer as the `outputPitch` is 0.");
    r = -70;
    goto error;
  }

  /* Setup some local aliases/helpers */
  out_ptr_y = (uint8_t*) outputDevicePtr;
  out_ptr_uv = out_ptr_y + (outputHeight * outputPitch);
  in_ptr_y = (uint8_t*) inputDevicePtr;
  in_ptr_uv = in_ptr_y + (inputHeight * inputPitch);
  scale_x = (float) inputWidth / (float) outputWidth;
  scale_y = (float) inputHeight / (float) outputHeight;

  /* ------------------------------------------- */
  /* Create the handles                          */
  /* ------------------------------------------- */

  tex_desc.filterMode = hipFilterModePoint;   /* Nearest neighbour scaling */
  tex_desc.readMode = hipReadModeElementType; /* No conversion from type is performed. See [this][2] documentation. */

  res_desc.resType = hipResourceTypePitch2D;
  res_desc.res.pitch2D.devPtr = inputDevicePtr;
  res_desc.res.pitch2D.width = inputWidth;
  res_desc.res.pitch2D.height = inputHeight;
  res_desc.res.pitch2D.pitchInBytes = inputPitch;

  /* Y-texture */
  res_desc.res.pitch2D.desc = hipCreateChannelDesc<uint8_t>();

  status = hipCreateTextureObject(&tex_y, &res_desc, &tex_desc, NULL);
  if (hipSuccess != status) {
    TRAE("Failed to create the texture object for the Y-plane.");
    r = -80;
    goto error;
  }

  /* UV-texture */
  res_desc.res.pitch2D.desc = hipCreateChannelDesc<uchar2>();
  res_desc.res.pitch2D.devPtr = in_ptr_uv;
  res_desc.res.pitch2D.width = (inputWidth / 2);  
  res_desc.res.pitch2D.height = (inputHeight / 2);
  
  status = hipCreateTextureObject(&tex_uv, &res_desc, &tex_desc, NULL);
  if (hipSuccess != status) {
    TRAE("Failed to create the texture object for the UV-plane.");
    r = -90;
    goto error;
  }

  /*
    
    NOTE: the [Resize.cu][1] implementation seems to run the
    kernel over all the output pixels of the Y-plane. As we fetch
    2 lines in the kernel this is not necessary. We only have to
    run the kernel over all of the UV-plane pixels. To make sure
    we only run the kernel over the UV-plane we use half the
    width and height of the output size.
    
    grid_size.x = (outputWidth + 31) / 32;
    grid_size.y = (outputHeight + 31) / 32;
    
  */
  grid_size.x = ((outputWidth / 2) + 15) / 16;
  grid_size.y = ((outputHeight / 2) + 15) / 16;
  grid_size.z = 1;

  /*
    
    Using 32,32,1 for the block size gave me pretty good results
    on a 2080x RTX (6.5us to resize from 1920x1080 to 960x540)
    but pretty bad results on a 1650gtx: 30us.

    Therefore I changed the block size to 16,16,1 and now I'm
    getting 2.8us (2080RTX) and 4.5us (1650GTX). I've also
    experimented with the `cudaOccupancyMacPotentialBlockSize()`
    function which gives me a number of 14 for the blocks size
    which turns out to be slower than using 16. Also, better
    occupancy doesn't always mean better performance, see
    [this][4].
    
   */
  block_size.x = 16;
  block_size.y = 16;
  block_size.z = 1;

  cuda_resize_nv12_kernel<<<grid_size, block_size>>>(
    tex_y,
    tex_uv,
    inputWidth,
    inputHeight,
    inputPitch,
    out_ptr_y,
    out_ptr_uv,
    outputWidth,
    outputHeight,
    outputPitch,
    scale_x,
    scale_y
  );

  status = hipGetLastError();
  if (hipSuccess != status) {
    TRAE("Failed to execute the kernel.");
    r = -100;
    goto error;
  }
  
 error:

  return r;
}

/* ------------------------------------------------------- */
